#include "gtest/gtest.h"
#include "reduce.h"
constexpr int input_size = 1 << 24;
constexpr value_t eps = 1e-4;

TEST(reduce, cpu) {
    value_t* input, * output;
	input = new value_t[input_size];
	output = new value_t;
	reduce_cpu_perf(input, output, input_size, 1);
	delete []input;
	delete output;
}

TEST(reduce, gpu_v1) {
    hipSetDevice(0);
    value_t* host_input, * host_output, * dev_input, * dev_output, * host_std;
    hipHostAlloc(&host_input, input_size * sizeof(value_t), hipHostMallocDefault);
    hipHostAlloc(&host_output, input_size * sizeof(value_t), hipHostMallocDefault);
    hipMalloc(&dev_input, input_size * sizeof(value_t));
    hipMalloc(&dev_output, input_size * sizeof(value_t));
    hipHostAlloc(&host_std, input_size * sizeof(value_t), hipHostMallocDefault);
	
    genData(host_input, input_size);
    hipMemcpy(dev_input, host_input, input_size * sizeof(value_t), hipMemcpyHostToDevice);

	int output_size = reduce_gpu_v1(dev_input, dev_output, input_size);
    hipMemcpy(host_output, dev_output, input_size * sizeof(value_t), hipMemcpyDeviceToHost);
	reduce_cpu(host_input, host_std, input_size, output_size);
	
	bool flag = compareData(host_output, host_std, output_size, eps);
	reduce_gpu_perf_v1(dev_input, dev_output, input_size);
	hipHostFree(host_input);
	hipHostFree(host_output);
	hipHostFree(host_std);
	hipFree(dev_input);
	hipFree(dev_output);
	EXPECT_TRUE(flag);
}

TEST(reduce, gpu_v2) {
    hipSetDevice(0);
    value_t* host_input, * host_output, * dev_input, * dev_output, * host_std;
    hipHostAlloc(&host_input, input_size * sizeof(value_t), hipHostMallocDefault);
    hipHostAlloc(&host_output, input_size * sizeof(value_t), hipHostMallocDefault);
    hipMalloc(&dev_input, input_size * sizeof(value_t));
    hipMalloc(&dev_output, input_size * sizeof(value_t));
    hipHostAlloc(&host_std, input_size * sizeof(value_t), hipHostMallocDefault);
	
    genData(host_input, input_size);
    hipMemcpy(dev_input, host_input, input_size * sizeof(value_t), hipMemcpyHostToDevice);

	int output_size = reduce_gpu_v2(dev_input, dev_output, input_size);
    hipMemcpy(host_output, dev_output, input_size * sizeof(value_t), hipMemcpyDeviceToHost);
	reduce_cpu(host_input, host_std, input_size, output_size);
	
	bool flag = compareData(host_output, host_std, output_size, eps);
	reduce_gpu_perf_v2(dev_input, dev_output, input_size);
	hipHostFree(host_input);
	hipHostFree(host_output);
	hipHostFree(host_std);
	hipFree(dev_input);
	hipFree(dev_output);
	EXPECT_TRUE(flag);
}

TEST(reduce, gpu_v3) {
    hipSetDevice(0);
    value_t* host_input, * host_output, * dev_input, * dev_output, * host_std;
    hipHostAlloc(&host_input, input_size * sizeof(value_t), hipHostMallocDefault);
    hipHostAlloc(&host_output, input_size * sizeof(value_t), hipHostMallocDefault);
    hipMalloc(&dev_input, input_size * sizeof(value_t));
    hipMalloc(&dev_output, input_size * sizeof(value_t));
    hipHostAlloc(&host_std, input_size * sizeof(value_t), hipHostMallocDefault);
	
    genData(host_input, input_size);
    hipMemcpy(dev_input, host_input, input_size * sizeof(value_t), hipMemcpyHostToDevice);

	int output_size = reduce_gpu_v3(dev_input, dev_output, input_size);
    hipMemcpy(host_output, dev_output, input_size * sizeof(value_t), hipMemcpyDeviceToHost);
	reduce_cpu(host_input, host_std, input_size, output_size);
	
	bool flag = compareData(host_output, host_std, output_size, eps);
	reduce_gpu_perf_v3(dev_input, dev_output, input_size);
	hipHostFree(host_input);
	hipHostFree(host_output);
	hipHostFree(host_std);
	hipFree(dev_input);
	hipFree(dev_output);
	EXPECT_TRUE(flag);
}

TEST(reduce, gpu_v4) {
    hipSetDevice(0);
    value_t* host_input, * host_output, * dev_input, * dev_output, * host_std;
    hipHostAlloc(&host_input, input_size * sizeof(value_t), hipHostMallocDefault);
    hipHostAlloc(&host_output, input_size * sizeof(value_t), hipHostMallocDefault);
    hipMalloc(&dev_input, input_size * sizeof(value_t));
    hipMalloc(&dev_output, input_size * sizeof(value_t));
    hipHostAlloc(&host_std, input_size * sizeof(value_t), hipHostMallocDefault);
	
    genData(host_input, input_size);
    hipMemcpy(dev_input, host_input, input_size * sizeof(value_t), hipMemcpyHostToDevice);

	int output_size = reduce_gpu_v4(dev_input, dev_output, input_size);
    hipMemcpy(host_output, dev_output, input_size * sizeof(value_t), hipMemcpyDeviceToHost);
	reduce_cpu(host_input, host_std, input_size, output_size);
	
	bool flag = compareData(host_output, host_std, output_size, eps);
	reduce_gpu_perf_v4(dev_input, dev_output, input_size);
	hipHostFree(host_input);
	hipHostFree(host_output);
	hipHostFree(host_std);
	hipFree(dev_input);
	hipFree(dev_output);
	EXPECT_TRUE(flag);
}
