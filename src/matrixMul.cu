#include "hip/hip_runtime.h"
#include "helper.h"
#include "matrixMul.h"

#define A(i,j) A[(i)*lda+(j)]
#define B(i,j) B[(i)*ldb+(j)]
#define C(i,j) C[(i)*ldc+(j)]
void print(value_t* A, int lda) {
    for (int i = 0; i < 10; i++) {
        for (int j = 0; j < 10; j++) {
            printf("%.2f ", A(i,j));
        }
        puts("");
    }
    puts("-----------------------------------------------------");
}
namespace matrixMul_1 {
    constexpr int  TILE = 1 << 5;
    __global__ void matrixMul(value_t* A, value_t* B, value_t* C, int lda, int ldb, int ldc) {
        __shared__ value_t shmemA[TILE][TILE + 1], shmemB[TILE][TILE + 1];
        int bx = blockIdx.x * TILE, by = blockIdx.y * TILE;
        int tx = threadIdx.x, ty = threadIdx.y;
        A = A + by * lda;
        B = B + bx;
        C = C + by * ldc + bx;
        value_t sum = 0;
        for (int i = 0; i < lda; i += TILE) {
            shmemA[tx][ty] = A(ty, tx + i);
            shmemB[ty][tx] = B(ty + i, tx);
            __syncthreads();
            for (int j = 0; j < TILE; j++) {
                sum += shmemA[j][ty] * shmemB[j][tx];
            }
            __syncthreads();
        }
        C(ty, tx) = sum;
    }
    void launch(value_t* dev_a, value_t* dev_b, value_t* dev_c, int N, int M, int K) {
        matrixMul << <dim3(M / TILE, N / TILE), dim3(TILE, TILE) >> > (dev_a, dev_b, dev_c, K, M, M);
    }
}
namespace matrixMul_2_1 {
    constexpr int  TILE = 1 << 7, TILE_K = 1 << 3, stride = 1 << 5, num = TILE / stride;
    __global__ void matrixMul(value_t* A, value_t* B, value_t* C, int lda, int ldb, int ldc) {
        __shared__ value_t shmemA[TILE_K][TILE + 1], shmemB[TILE_K][TILE + 1];
        int bx = blockIdx.x * TILE, by = blockIdx.y * TILE;
        int tx = threadIdx.x, ty = threadIdx.y, tid = threadIdx.y * blockDim.x + threadIdx.x;
        int shax = tid % TILE_K, shay = tid / TILE_K;
        int shbx = tid % TILE, shby = tid / TILE;
        A = A + by * lda;
        B = B + bx;
        C = C + by * ldc + bx;
        value_t regA[num];
        value_t regB[num];
        value_t sum[num][num] = {};
        for (int i = 0; i < lda; i += TILE_K) {
            shmemA[shax][shay] = A(shay, shax + i);
            shmemB[shby][shbx] = B(shby + i, shbx);
            __syncthreads();
            for (int j = 0; j < TILE_K; j++) {
                for (int k = 0; k < num; k++) {
                    regA[k] = shmemA[j][ty + k * stride];
                    regB[k] = shmemB[j][tx + k * stride];
                }
                for (int k1 = 0; k1 < num; k1++) {
                    for (int k2 = 0; k2 < num; k2++) {
                        sum[k1][k2] += regA[k1] * regB[k2];
                    }
                }
            }
            __syncthreads();
        }
        for (int k1 = 0; k1 < num; k1++) {
            for (int k2 = 0; k2 < num; k2++) {
                C(ty + k1 * stride, tx + k2 * stride) = sum[k1][k2];
            }
        }
    }
    void launch(value_t* dev_a, value_t* dev_b, value_t* dev_c, int N, int M, int K) {
        matrixMul << <dim3(M / TILE, N / TILE), dim3(stride, stride) >> > (dev_a, dev_b, dev_c, K, M, M);
    }
}
namespace matrixMul_2_2 {
    constexpr int  TILE = 1 << 7, TILE_K = 1 << 3, num = 1 << 2;
    __global__ void matrixMul(value_t* A, value_t* B, value_t* C, int lda, int ldb, int ldc) {
        __shared__ value_t shmemA[TILE_K][TILE], shmemB[TILE_K][TILE];
        int bx = blockIdx.x * TILE, by = blockIdx.y * TILE;
        int tx = threadIdx.x * num, ty = threadIdx.y * num, tid = threadIdx.y * blockDim.x + threadIdx.x;
        int shax = tid % TILE_K, shay = tid / TILE_K;
        int shbx = tid % TILE, shby = tid / TILE;
        A = A + by * lda;
        B = B + bx;
        C = C + by * ldc + bx;
        value_t regA[num];
        value_t regB[num];
        value_t sum[num][num] = {};
        for (int i = 0; i < lda; i += TILE_K) {
            shmemA[shax][shay] = A(shay, shax + i);
            shmemB[shby][shbx] = B(shby + i, shbx);
            __syncthreads();
            for (int j = 0; j < TILE_K; j++) {
                for (int k = 0; k < num; k++) {
                    regA[k] = shmemA[j][ty + k];
                    regB[k] = shmemB[j][tx + k];
                }
                for (int k1 = 0; k1 < num; k1++) {
                    for (int k2 = 0; k2 < num; k2++) {
                        sum[k1][k2] += regA[k1] * regB[k2];
                    }
                }
            }
            __syncthreads();
        }
        for (int k1 = 0; k1 < num; k1++) {
            for (int k2 = 0; k2 < num; k2++) {
                C(ty + k1, tx + k2) = sum[k1][k2];
            }
        }
    }
    void launch(value_t* dev_a, value_t* dev_b, value_t* dev_c, int N, int M, int K) {
        matrixMul << <dim3(M / TILE, N / TILE), dim3(TILE / num, TILE / num) >> > (dev_a, dev_b, dev_c, K, M, M);
    }
}
namespace matrixMul_3 {
    constexpr int  TILE = 1 << 7, TILE_K = 1 << 3, num = 1 << 2;
    __global__ void matrixMul(value_t* A, value_t* B, value_t* C, int lda, int ldb, int ldc) {
        __shared__ float4 shmemA[TILE_K][TILE >> 2], shmemB[TILE_K][TILE >> 2];
        int bx = blockIdx.x * TILE, by = blockIdx.y * TILE;
        int tx = threadIdx.x * num, ty = threadIdx.y * num, tid = threadIdx.y * blockDim.x + threadIdx.x;
        int shax = tid % TILE_K, shay = tid / TILE_K;
        int shbx = tid % TILE, shby = tid / TILE;
        A = A + by * lda;
        B = B + bx;
        C = C + by * ldc + bx;
        float4 regA;
        float4 regB;
        float sum[num][num] = {};
        float* pa = (float*)shmemA, * pb = (float*)shmemB;
        for (int i = 0; i < lda; i += TILE_K) {
            pa[shax * TILE + shay] = A(shay, shax + i);
            pb[shby * TILE + shbx] = B(shby + i, shbx);
            __syncthreads();
            for (int j = 0; j < TILE_K; j++) {
                regA = shmemA[j][ty >> 2];
                regB = shmemB[j][tx >> 2];
                sum[0][0] += regA.x * regB.x;
                sum[0][1] += regA.x * regB.y;
                sum[0][2] += regA.x * regB.z;
                sum[0][3] += regA.x * regB.w;
                sum[1][0] += regA.y * regB.x;
                sum[1][1] += regA.y * regB.y;
                sum[1][2] += regA.y * regB.z;
                sum[1][3] += regA.y * regB.w;
                sum[2][0] += regA.z * regB.x;
                sum[2][1] += regA.z * regB.y;
                sum[2][2] += regA.z * regB.z;
                sum[2][3] += regA.z * regB.w;
                sum[3][0] += regA.w * regB.x;
                sum[3][1] += regA.w * regB.y;
                sum[3][2] += regA.w * regB.z;
                sum[3][3] += regA.w * regB.w;
            }
            __syncthreads();
        }
        for (int k1 = 0; k1 < num; k1++) {
            for (int k2 = 0; k2 < num; k2++) {
                C(ty + k1, tx + k2) = sum[k1][k2];
            }
        }
    }
    void launch(value_t* dev_a, value_t* dev_b, value_t* dev_c, int N, int M, int K) {
        matrixMul << <dim3(M / TILE, N / TILE), dim3(TILE / num, TILE / num) >> > (dev_a, dev_b, dev_c, K, M, M);
    }
}
void testMatrixMul()
{
    const int N = 1 << 12, M = 1 << 12, K = 1 << 12;
    const value_t eps = 5e-6;
    value_t* host_a, * host_b, * host_c, * host_std, * dev_a, * dev_b, * dev_c;
    hipSetDevice(0);
    hipHostAlloc(&host_a, N * K * sizeof(value_t), hipHostMallocDefault);
    hipHostAlloc(&host_b, K * M * sizeof(value_t), hipHostMallocDefault);
    hipHostAlloc(&host_c, N * M * sizeof(value_t), hipHostMallocDefault);
    hipHostAlloc(&host_std, N * M * sizeof(value_t), hipHostMallocDefault);
    hipMalloc(&dev_a, N * K * sizeof(value_t));
    hipMalloc(&dev_b, K * M * sizeof(value_t));
    hipMalloc(&dev_c, N * M * sizeof(value_t));
    genData(host_a, N * K);
    genData(host_b, K * M);

    hipMemcpy(dev_a, host_a, N * K * sizeof(value_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, K * M * sizeof(value_t), hipMemcpyHostToDevice);

    TIMERSTART(matrixMul_std);
    hipMemset(dev_c, 0, N * M * sizeof(value_t));
    value_t alpha = 1;
    value_t beta = 0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, dev_b, M, dev_a, K, &beta, dev_c, M); CUERR;
    TIMERSTOP(matrixMul_std);
    hipMemcpy(host_std, dev_c, N * M * sizeof(value_t), hipMemcpyDeviceToHost);

    TIMERSTART(matrixMul_1);
    hipMemset(dev_c, 0, N * M * sizeof(value_t));
    matrixMul_1::launch(dev_a, dev_b, dev_c, N, M, K); CUERR;
    TIMERSTOP(matrixMul_1);
    hipMemcpy(host_c, dev_c, N * M * sizeof(value_t), hipMemcpyDeviceToHost);
    compareData(host_std, host_c, N * M, eps);

    TIMERSTART(matrixMul_2_1);
    hipMemset(dev_c, 0, N * M * sizeof(value_t));
    matrixMul_2_1::launch(dev_a, dev_b, dev_c, N, M, K); CUERR;
    TIMERSTOP(matrixMul_2_1);
    hipMemcpy(host_c, dev_c, N * M * sizeof(value_t), hipMemcpyDeviceToHost);
    compareData(host_std, host_c, N * M, eps);

    TIMERSTART(matrixMul_2_2);
    hipMemset(dev_c, 0, N * M * sizeof(value_t));
    matrixMul_2_2::launch(dev_a, dev_b, dev_c, N, M, K); CUERR;
    TIMERSTOP(matrixMul_2_2);
    hipMemcpy(host_c, dev_c, N * M * sizeof(value_t), hipMemcpyDeviceToHost);
    compareData(host_std, host_c, N * M, eps);

    TIMERSTART(matrixMul_3);
    hipMemset(dev_c, 0, N * M * sizeof(value_t));
    matrixMul_3::launch(dev_a, dev_b, dev_c, N, M, K); CUERR;
    TIMERSTOP(matrixMul_3);
    hipMemcpy(host_c, dev_c, N * M * sizeof(value_t), hipMemcpyDeviceToHost);
    compareData(host_std, host_c, N * M, eps);

    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}
void matrixMul(value_t* dev_a, value_t* dev_b, value_t* dev_c, int N, int M, int K) {
    matrixMul_3::launch(dev_a, dev_b, dev_c, N, M, K);
}
